#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  gpu.cu
 *
 *    Description:  GPU-dependent code for Mandelbrot generator
 *
 *        Version:  1.0
 *        Created:  06/11/2019 09:40:39 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  YOUR NAME (), 
 *   Organization:  
 *
 * =====================================================================================
 */
#include "Mandelbrot.h"
#include <iostream>

__host__ __device__
int32_t iterations(Point coords, uint32_t threshold) {
	Real z_r = 0;
	Real z_i = 0;

	Real old_z_r = 0;
	Real old_z_i = 0;

	auto coords_r = coords.real();
	auto coords_i = coords.imag();

	for (int i = 0; i < threshold; i++) {
		old_z_r = z_r;
		old_z_i = z_i;

		z_r = old_z_r * old_z_r - old_z_i * old_z_i + coords_r;
		z_i = 2 * old_z_r * old_z_i + coords_i;

		if (sqrt(z_r * z_r + z_i * z_i) > 2.0) return i;
	}
	return -1;
}

__host__ __device__
Point coord_map(Bounds bounds, Coord size, Coord location) {
	Real x1 = bounds.first.real();
	Real y1 = bounds.first.imag();
	Real x2 = bounds.second.real();
	Real y2 = bounds.second.imag();

	Real x_dim = x2 - x1;
	Real y_dim = y2 - y1;

	Real x_coord = x1 + x_dim * (((double) location.first) / size.first);
	Real y_coord = y1 + y_dim * (((double) location.second) / size.second);

	return Point(x_coord, y_coord);
}

__host__
Bounds frame_bounds(
		Bounds bounds, Point center,
		int frame, Real reduction
		) {
	Real width  = (bounds.second.real() - bounds.first.real()) * pow((1 - reduction), frame);
	Real height = (bounds.second.imag() - bounds.first.imag()) * pow((1 - reduction), frame);

	Real x1 = center.real() - (width / 2);
	Real x2 = center.real() + (width / 2);
	Real y1 = center.imag() - (height / 2);
	Real y2 = center.imag() + (height / 2);
	return Bounds(Point(x1, y1), Point(x2, y2));
}

__global__
void populate_row(int *p, uint32_t y, uint32_t width, uint32_t height, Bounds bounds, uint32_t threshold) {

	for (auto x = 0; x < width; x++) {
		Point coords = coord_map(bounds, Coord(width, height), Coord(x, y));
		p[x] = iterations(coords, threshold);
	}
}

__host__
void generate_image(Image &image, Bounds bounds, uint32_t threshold) {
	auto width = image.columns();
	auto height = image.rows();

	image.modifyImage();

	Pixels pixel_cache(image);
	auto pixels = pixel_cache.get(0, 0, width, height);

	Color black(0, 0, 0, 0);
	auto color_scale = ((Real) MaxRGB / threshold);

	int32_t *p;

	for (auto y = 0; y < height; y++) {
		hipMallocManaged(&p, width * sizeof(int32_t));

		populate_row<<<1, 1>>>(p, y, width, height, bounds, threshold);

		for (auto i = 0; i < width; i++) {
			int32_t depth = p[i];
			if (depth == -1) {
				*(pixels + width * y + i) = black;
			} else {
				*(pixels + width * y + i) = Color(
						color_scale * depth,
						color_scale * depth / 2.0,
						MaxRGB - color_scale * depth);
			}
		}

		hipFree(p);
	}

	hipDeviceSynchronize();

	pixel_cache.sync();
}

